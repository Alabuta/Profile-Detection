#include "hip/hip_runtime.h"
#include <algorithm>
#include "cudaImpls.h"

#if __USE_GPGPU__

#if _DEBUG
#   pragma comment(lib, "opencv_core320d.lib")
#   pragma comment(lib, "opencv_cudafilters320d.lib")
#   pragma comment(lib, "opencv_cudaimgproc320d.lib")
#   pragma comment(lib, "opencv_highgui320d.lib")
#   pragma comment(lib, "opencv_imgcodecs320d.lib")
#   pragma comment(lib, "opencv_imgproc320d.lib")
#elif NDEBUG
#   pragma comment(lib, "opencv_core320.lib")
#   pragma comment(lib, "opencv_cudafilters320.lib")
#   pragma comment(lib, "opencv_cudaimgproc320.lib")
#   pragma comment(lib, "opencv_highgui320.lib")
#   pragma comment(lib, "opencv_imgcodecs320.lib")
#   pragma comment(lib, "opencv_imgproc320.lib")
#endif

#ifdef __HIPCC__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif

#pragma comment(lib, "cuda.lib")
#pragma comment(lib, "cudart.lib")

using namespace std;

__global__ void inRange_kernel(const cv::cuda::PtrStepSz<uchar3> src, cv::cuda::PtrStepSzb dst,
    int lbc0, int ubc0, int lbc1, int ubc1, int lbc2, int ubc2)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= src.cols || y >= src.rows) return;

    uchar3 v = src(y, x);

    if (v.x >= lbc0 && v.x <= ubc0 && v.y >= lbc1 && v.y <= ubc1 && v.z >= lbc2 && v.z <= ubc2)
        dst(y, x) = 255;
    else
        dst(y, x) = 0;
}

void inRange_gpu(cv::cuda::GpuMat &src, cv::Scalar &lowerb, cv::Scalar &upperb, cv::cuda::GpuMat &dst)
{
    const int m = 32;
    int numRows = src.rows, numCols = src.cols;
    if (numRows == 0 || numCols == 0) return;

    const dim3 gridSize(ceil((float)numCols / m), ceil((float)numRows / m), 1);
    const dim3 blockSize(m, m, 1);

    inRange_kernel KERNEL_ARGS2(gridSize, blockSize) (src, dst, lowerb[0], upperb[0], lowerb[1], upperb[1], lowerb[2], upperb[2]);
}
#endif